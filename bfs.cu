#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <vector>
#include <queue>

__global__ void random_graph_initializer(int *src,int *dest,int numEdges,int numNodes,unsigned int seed){
 int threadId=blockIdx.x*blockDim.x + threadIdx.x;
 if(threadId<numEdges){
    hiprandState state;
    hiprand_init(seed,threadId,0,&state);
    src[threadId]=hiprand(&state)%numNodes;
    dest[threadId]=hiprand(&state)%numNodes;
 }   
}

__global__ void countEdges(int *src,int *rowPtr,int numEdges){
 int threadId=blockIdx.x*blockDim.x + threadIdx.x;
 if(threadId<numEdges){
    atomicAdd(&rowPtr[src[threadId]+1],1);
 }
}

__global__ void bfs_kernel(int *rowPtr,int *colIdx,int *visited,int *frontier,int *next_frontier,int numNodes,int *done){
 int threadId=blockIdx.x*blockDim.x + threadIdx.x;
 if(threadId<numNodes && frontier[threadId]){
    frontier[threadId]=0;
    int st=rowPtr[threadId];
    int end=rowPtr[threadId+1];
    for(int i=st;i<end;i++){
        int v=colIdx[i];
        if(!visited[v]){
            if(atomicCAS(&visited[v],0,1)==0){
                next_frontier[v]=1;
                *done=0;
            }
        }
    }
 }
}

void bfs_cpu(std::vector<int>&rowPtr,std::vector<int>&colIdx,std::vector<int>&visited,int start){
 std::queue<int>q;
 visited[start]=1;
 q.push(start);
 while(!q.empty()){
  int u=q.front();
  q.pop();
  for(int i=rowPtr[u];i<rowPtr[u+1];i++){
   int v=colIdx[i];
   if(!visited[v]){
    visited[v]=1;
    q.push(v);
   }
  }
 }
}

int main(){
 srand((unsigned)time(NULL));
 unsigned int seed=(unsigned)time(NULL);
 int numNodes=10000;
 int numEdges=50000;
 printf("Random graph: %d nodes, %d edges\n",numNodes,numEdges);

 int *d_src,*d_dest,*d_rowPtr,*d_colIdx;
 hipMalloc(&d_src,numEdges*sizeof(int));
 hipMalloc(&d_dest,numEdges*sizeof(int));
 hipMalloc(&d_rowPtr,(numNodes+1)*sizeof(int));
 hipMalloc(&d_colIdx,numEdges*sizeof(int));
 hipMemset(d_rowPtr,0,(numNodes+1)*sizeof(int));

 int threads=256;
 int blocks=(numEdges+threads-1)/threads;

 random_graph_initializer<<<blocks,threads>>>(d_src,d_dest,numEdges,numNodes,seed);
 hipDeviceSynchronize();
 countEdges<<<blocks,threads>>>(d_src,d_rowPtr,numEdges);
 hipDeviceSynchronize();

 int *h_rowPtr=(int*)malloc((numNodes+1)*sizeof(int));
 hipMemcpy(h_rowPtr,d_rowPtr,(numNodes+1)*sizeof(int),hipMemcpyDeviceToHost);
 for(int i=1;i<=numNodes;i++){
    h_rowPtr[i]+=h_rowPtr[i-1];
 }
 hipMemcpy(d_rowPtr,h_rowPtr,(numNodes+1)*sizeof(int),hipMemcpyHostToDevice);

 int *h_src=(int*)malloc(numEdges*sizeof(int));
 int *h_dest=(int*)malloc(numEdges*sizeof(int));
 hipMemcpy(h_src,d_src,numEdges*sizeof(int),hipMemcpyDeviceToHost);
 hipMemcpy(h_dest,d_dest,numEdges*sizeof(int),hipMemcpyDeviceToHost);

 int *h_colIdx=(int*)malloc(numEdges*sizeof(int));
 int *pos=(int*)calloc(numNodes,sizeof(int));
 for(int i=0;i<numEdges;i++){
    int u=h_src[i];
    int idx=h_rowPtr[u]+pos[u]++;
    h_colIdx[idx]=h_dest[i];
 }
 hipMemcpy(d_colIdx,h_colIdx,numEdges*sizeof(int),hipMemcpyHostToDevice);

 std::vector<int>rowPtr(h_rowPtr,h_rowPtr+numNodes+1);
 std::vector<int>colIdx(h_colIdx,h_colIdx+numEdges);
 std::vector<int>visited_cpu(numNodes,0);

 auto start_cpu=std::chrono::high_resolution_clock::now();
 bfs_cpu(rowPtr,colIdx,visited_cpu,0);
 auto end_cpu=std::chrono::high_resolution_clock::now();
 double cpu_ms=std::chrono::duration<double,std::milli>(end_cpu-start_cpu).count();

 int *d_visited,*d_frontier,*d_next_frontier,*d_done;
 hipMalloc(&d_visited,numNodes*sizeof(int));
 hipMalloc(&d_frontier,numNodes*sizeof(int));
 hipMalloc(&d_next_frontier,numNodes*sizeof(int));
 hipMalloc(&d_done,sizeof(int));
 hipMemset(d_visited,0,numNodes*sizeof(int));
 hipMemset(d_frontier,0,numNodes*sizeof(int));
 hipMemset(d_next_frontier,0,numNodes*sizeof(int));
 int one=1;
 hipMemcpy(d_visited,&one,sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(d_frontier,&one,sizeof(int),hipMemcpyHostToDevice);

 threads=256;
 blocks=(numNodes+threads-1)/threads;
 int h_done;

 hipEvent_t start,stop;
 hipEventCreate(&start);
 hipEventCreate(&stop);
 hipEventRecord(start);
 do{
  h_done=1;
  hipMemcpy(d_done,&h_done,sizeof(int),hipMemcpyHostToDevice);
  bfs_kernel<<<blocks,threads>>>(d_rowPtr,d_colIdx,d_visited,d_frontier,d_next_frontier,numNodes,d_done);
  hipDeviceSynchronize();
  hipMemcpy(&h_done,d_done,sizeof(int),hipMemcpyDeviceToHost);
  hipMemcpy(d_frontier,d_next_frontier,numNodes*sizeof(int),hipMemcpyDeviceToDevice);
  hipMemset(d_next_frontier,0,numNodes*sizeof(int));
 }while(h_done==0);
 hipEventRecord(stop);
 hipEventSynchronize(stop);
 float gpu_ms=0;
 hipEventElapsedTime(&gpu_ms,start,stop);

 printf("CPU BFS time: %.2f ms\n",cpu_ms);
 printf("GPU BFS time: %.2f ms\n",gpu_ms);
 printf("Speedup: %.2fx\n",cpu_ms/gpu_ms);

 free(h_rowPtr);
 free(h_src);
 free(h_dest);
 free(h_colIdx);
 free(pos);
 hipFree(d_src);
 hipFree(d_dest);
 hipFree(d_rowPtr);
 hipFree(d_colIdx);
 hipFree(d_visited);
 hipFree(d_frontier);
 hipFree(d_next_frontier);
 hipFree(d_done);
 return 0;
}
