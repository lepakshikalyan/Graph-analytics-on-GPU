#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <vector>

__global__ void random_graph_initializer(int *src,int *dest,int numEdges,int numNodes,unsigned int seed){
 int threadId=blockIdx.x*blockDim.x + threadIdx.x;
 if(threadId<numEdges){
    hiprandState state;
    hiprand_init(seed,threadId,0,&state);
    src[threadId]=hiprand(&state)%numNodes;
    dest[threadId]=hiprand(&state)%numNodes;
 }   
}

__global__ void countEdges(int *src,int *rowPtr,int numEdges){
 int threadId=blockIdx.x*blockDim.x + threadIdx.x;
 if(threadId<numEdges){
    atomicAdd(&rowPtr[src[threadId]+1],1);
 }
}

__global__ void pagerankKernel(int *rowPtr,int *colIdx,float *rankOld,float *rankNew,int numNodes,float damping){
 int threadId=blockIdx.x*blockDim.x + threadIdx.x;
 if(threadId<numNodes){
    float sum=0.0f;
    int st=rowPtr[threadId];
    int end=rowPtr[threadId+1];
    for(int i=st;i<end;i++){
        int v=colIdx[i];
        int outDegree=rowPtr[v+1]-rowPtr[v];
        if(outDegree>0){
            sum+=(rankOld[v]/outDegree);
        }
    }
    rankNew[threadId]=(1.0f-damping)/numNodes+(damping*sum);
 }
}

void pagerank_cpu(std::vector<int>&rowPtr,std::vector<int>&colIdx,std::vector<float>&rank,int numNodes,float damping,int max_iters){
 std::vector<float>newRank(numNodes,0.0f);
 for(int iter=0;iter<max_iters;iter++){
  for(int i=0;i<numNodes;i++){
   float sum=0.0f;
   for(int j=rowPtr[i];j<rowPtr[i+1];j++){
    int v=colIdx[j];
    int outdeg=rowPtr[v+1]-rowPtr[v];
    if(outdeg>0) sum+=rank[v]/outdeg;
   }
   newRank[i]=(1.0f-damping)/numNodes+damping*sum;
  }
  rank.swap(newRank);
 }
}

int main(){
 srand((unsigned)time(NULL));
 unsigned int seed=(unsigned)time(NULL);
 int numNodes=10000;
 int numEdges=50000;
 printf("Random graph: %d nodes, %d edges\n",numNodes,numEdges);
 
 int *d_src,*d_dest,*d_rowPtr,*d_colIdx;
 hipMalloc(&d_src,numEdges*sizeof(int));
 hipMalloc(&d_dest,numEdges*sizeof(int));
 hipMalloc(&d_rowPtr,(numNodes+1)*sizeof(int));
 hipMalloc(&d_colIdx,numEdges*sizeof(int));
 hipMemset(d_rowPtr,0,(numNodes+1)*sizeof(int));

 int threads=256;
 int blocks=(numEdges+threads-1)/threads;

 random_graph_initializer<<<blocks,threads>>>(d_src,d_dest,numEdges,numNodes,seed);
 hipDeviceSynchronize();
 countEdges<<<blocks,threads>>>(d_src,d_rowPtr,numEdges);
 hipDeviceSynchronize();

 int *h_rowPtr=(int*)malloc((numNodes+1)*sizeof(int));
 hipMemcpy(h_rowPtr,d_rowPtr,(numNodes+1)*sizeof(int),hipMemcpyDeviceToHost);
 for(int i=1;i<=numNodes;i++){
    h_rowPtr[i]+=h_rowPtr[i-1];
 }
 hipMemcpy(d_rowPtr,h_rowPtr,(numNodes+1)*sizeof(int),hipMemcpyHostToDevice);
 
 int *h_src=(int*)malloc(numEdges*sizeof(int));
 int *h_dest=(int*)malloc(numEdges*sizeof(int));
 hipMemcpy(h_src,d_src,numEdges*sizeof(int),hipMemcpyDeviceToHost);
 hipMemcpy(h_dest,d_dest,numEdges*sizeof(int),hipMemcpyDeviceToHost);
 
 int *h_colIdx=(int*)malloc(numEdges*sizeof(int));
 int *pos=(int*)calloc(numNodes,sizeof(int));
 for(int i=0;i<numEdges;i++){
    int v=h_dest[i];
    int idx=h_rowPtr[v]+pos[v]++;
    h_colIdx[idx]=h_src[i];
 }
 hipMemcpy(d_colIdx,h_colIdx,numEdges*sizeof(int),hipMemcpyHostToDevice);
 
 float *d_rankOld,*d_rankNew;
 hipMalloc(&d_rankOld,numNodes*sizeof(float));
 hipMalloc(&d_rankNew,numNodes*sizeof(float));
 
 float *h_rank=(float*)malloc(numNodes*sizeof(float));
 for(int i=0;i<numNodes;i++) h_rank[i]=1.0f/numNodes;
 hipMemcpy(d_rankOld,h_rank,numNodes*sizeof(float),hipMemcpyHostToDevice);
 
 threads=256;
 blocks=(numNodes+threads-1)/threads;
 float damping=0.85f;
 int max_iters=20;

 std::vector<int>rowPtr(h_rowPtr,h_rowPtr+numNodes+1);
 std::vector<int>colIdx(h_colIdx,h_colIdx+numEdges);
 std::vector<float>rank_cpu(numNodes,1.0f/numNodes);

 auto start_cpu=std::chrono::high_resolution_clock::now();
 pagerank_cpu(rowPtr,colIdx,rank_cpu,numNodes,damping,max_iters);
 auto end_cpu=std::chrono::high_resolution_clock::now();
 double cpu_ms=std::chrono::duration<double,std::milli>(end_cpu-start_cpu).count();

 hipEvent_t start,stop;
 hipEventCreate(&start);
 hipEventCreate(&stop);
 hipEventRecord(start);
 for(int iter=0;iter<max_iters;iter++){
    pagerankKernel<<<blocks,threads>>>(d_rowPtr,d_colIdx,d_rankOld,d_rankNew,numNodes,damping);
    hipDeviceSynchronize();
    float *tmp=d_rankOld;
    d_rankOld=d_rankNew;
    d_rankNew=tmp;
 }
 hipEventRecord(stop);
 hipEventSynchronize(stop);
 float gpu_ms=0;
 hipEventElapsedTime(&gpu_ms,start,stop);

 hipMemcpy(h_rank,d_rankOld,numNodes*sizeof(float),hipMemcpyDeviceToHost);
 
 printf("CPU PR time: %.2f ms\n",cpu_ms);
 printf("GPU PR time: %.2f ms\n",gpu_ms);
 printf("Speedup: %.2fx\n",cpu_ms/gpu_ms);

 free(h_rowPtr);
 free(h_src);
 free(h_dest);
 free(h_colIdx);
 free(pos);
 free(h_rank);
 hipFree(d_src);
 hipFree(d_dest);
 hipFree(d_rowPtr);
 hipFree(d_colIdx);
 hipFree(d_rankOld);
 hipFree(d_rankNew);
 return 0;
}
